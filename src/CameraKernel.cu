#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include "Camera.cuh"
#include "CudaDeviceVec.cuh"
#include "CudaSerializable.cuh"
#include "Image.h"
#include "Random.cuh"
#include "World.cuh"

using namespace PPCast;

__global__ void renderInit(
    hiprandState *randomState, int width, int height,
    uint32_t seed
) {
    // Compute pixel index
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    if ((x >= width) || (y >= height)) return;

    // Each thread gets the same seed with a different sequence number and no offset
    const int pixelIndex = y * width + x;
    hiprand_init(seed, pixelIndex, 0, &randomState[pixelIndex]);
}

__global__ void renderImageGPUKernel(
    float3 *frameBuffer, int width, int height,
    VectorRef<Material>* materials,
    VectorRef<GeometryNode>* geometry,
    Camera* camera, hiprandState *randomState
) {
    // Compute pixel index
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    if ((x >= width) || (y >= height)) return;
    const int pixelIndex = y * width + x;

    // Compute VectorRef data pointers
    materials->data = reinterpret_cast<Material*>    (reinterpret_cast<VectorRef<Material>*>(materials) + 1);
    geometry ->data = reinterpret_cast<GeometryNode*>(reinterpret_cast<VectorRef<GeometryNode>*>(geometry) + 1);
    World world(*materials, *geometry);

    // Generate ray
    RandomState rs(&randomState[pixelIndex]);
    glm::vec3 colour = camera->renderPixel(x, y, world, rs);

    // Write colour to framebuffer
    frameBuffer[pixelIndex] = {colour.x, colour.y, colour.z};
}

bool Camera::renderImageGPU(Image& image, const World& world) const {
    // Allocate device buffers
    unsigned int numPixels = width * height;
    CudaDeviceVec<float3>                  d_frameBuffer(numPixels);
    CudaDeviceVec<hiprandState>             d_randState  (numPixels);
    CudaDeviceBox<VectorRef<Material>>     d_materials  (world.materials);
    CudaDeviceBox<VectorRef<GeometryNode>> d_geometry   (world.geometry);
    CudaDeviceVec<Camera>                  d_camera     (*this);

    // Upload data to device
    d_materials.copyToDevice();
    d_geometry .copyToDevice();
    d_camera   .copyToDevice();

    // Compute thread block dimensions
    int tx = 8;
    int ty = 8;
    dim3 blocks(width / tx + 1, height / ty + 1);
    dim3 threads(tx,ty);

    // Initialize
    renderInit<<<blocks, threads>>>(
        d_randState.get(), width, height, seed
    );

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Render image
    renderImageGPUKernel<<<blocks, threads>>>(
        d_frameBuffer.get(), width, height,
        d_materials.get(), d_geometry.get(),
        d_camera.get(), d_randState.get()
    );

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Copy image back to host
    d_frameBuffer.copyToHost(reinterpret_cast<float3*>(image.data()));

    return true;
}