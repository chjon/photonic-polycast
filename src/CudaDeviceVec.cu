#include "hip/hip_runtime.h"
#include <cstdio>
#include "CudaDeviceVec.cuh"

using namespace PPCast;

template <typename T>
CudaDeviceVec<T>::CudaDeviceVec(unsigned int size)
    : m_data(nullptr)
    , m_size(size)
{
    if (hipMalloc((void**)&m_data, sizeof(T) * m_size) != hipSuccess) m_size = 0;
}

template <typename T>
CudaDeviceVec<T>::CudaDeviceVec(const std::vector<T>& data)
    : m_data(nullptr)
    , m_size(0)
{
    if (hipMalloc((void**)&m_data, sizeof(T) * m_size) != hipSuccess) return;

    if (hipMemcpy(m_data, data.data(), sizeof(T) * m_size, hipMemcpyHostToDevice) == hipSuccess) {
        m_size = data.size();
    } else {
        hipFree(m_data);
        m_data = nullptr;
    }
}

template <typename T>
CudaDeviceVec<T>::~CudaDeviceVec() {
    if (m_size) hipFree(m_data);
}

template <typename T>
bool CudaDeviceVec<T>::copyToDevice(const T* hostData, unsigned int size) {
    return hipMemcpy(m_data, hostData, sizeof(T) * size, hipMemcpyHostToDevice) == hipSuccess;
}

template <typename T>
bool CudaDeviceVec<T>::copyToHost(T* hostData, unsigned int size) const {
    return hipMemcpy(hostData, m_data, sizeof(T) * size, hipMemcpyDeviceToHost) == hipSuccess;
}

// Explicit class initialization
template class CudaDeviceVec<float>;
template class CudaDeviceVec<float2>;
template class CudaDeviceVec<float3>;
template class CudaDeviceVec<float4>;