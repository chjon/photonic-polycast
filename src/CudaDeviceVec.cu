#include <cstdio>
#include "CudaDeviceVec.h"

using namespace PPCast;

template <typename T>
CudaDeviceVec<T>::CudaDeviceVec(unsigned int size)
    : m_data(nullptr)
    , m_size(size)
{
    if (hipMalloc((void**)&m_data, sizeof(T) * m_size) != hipSuccess) m_size = 0;
}

template <typename T>
CudaDeviceVec<T>::~CudaDeviceVec() {
    if (m_size) hipFree(m_data);
}

template <typename T>
bool CudaDeviceVec<T>::copyToDevice(const T* hostData, unsigned int size) {
    return hipMemcpy(m_data, hostData, sizeof(T) * size, hipMemcpyHostToDevice) == hipSuccess;
}

template <typename T>
bool CudaDeviceVec<T>::copyToHost(T* hostData, unsigned int size) const {
    return hipMemcpy(hostData, m_data, sizeof(T) * size, hipMemcpyDeviceToHost) == hipSuccess;
}

// Explicit class initialization
template class CudaDeviceVec<float>;